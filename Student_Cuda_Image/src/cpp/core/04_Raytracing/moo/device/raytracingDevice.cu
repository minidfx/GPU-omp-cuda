#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "IndiceTools.h"
#include "cudaTools.h"
#include "Device.h"
#include "RayTracingMath.h"
#include "Sphere.h"


/*----------------------------------------------------------------------*\
|*			Declaration                     *|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Imported	    *|
 \*-------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheres, int n, float t);

__global__ void rayTracingSM(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheres, int n, float t);

/*--------------------------------------*\
|*		Private			*|
 \*-------------------------------------*/

static __device__ void copyGMtoSM(Sphere* tabSpheresSM, Sphere* ptrDevSpheresGM, int n);

static __device__ void work(int w, int h, int n, float t, Sphere* ptrSpheres, uchar4* ptrDevPixels);

/*----------------------------------------------------------------------*\
|*			Implementation                  *|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
|*		Public			*|
 \*-------------------------------------*/

__global__
void rayTracingSM(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheresGM, int n, float t)
    {
    extern __shared__ Sphere
    tabSpheresSM[];
    copyGMtoSM(tabSpheresSM, ptrDevSpheresGM, n);
    __syncthreads();
    work(w, h, n, t, tabSpheresSM, ptrDevPixels);
    }

__global__
void rayTracing(uchar4* ptrDevPixels, int w, int h, Sphere* ptrDevSpheresGM, int n, float t)
    {
    work(w, h, n, t, ptrDevSpheresGM, ptrDevPixels);
    }

/*--------------------------------------*\
|*		Private			*|
 \*-------------------------------------*/

__device__
void work(int w, int h, int n, float t, Sphere* ptrDevSpheres, uchar4* ptrDevPixels)
    {
    RayTracingMath rayTracingMath = RayTracingMath(ptrDevSpheres, n, t);
    const int WH = w * h;
    const int NB_THREAD = Indice2D::nbThread();
    const int TID = Indice2D::tid();
    int s = TID;
    int i;
    int j;
    uchar4 color;
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[
	// rayTracingMath.color(i, j, t, ptrDevPixels[s]);
	rayTracingMath.colorIJ(&color, i, j);  // update color
	ptrDevPixels[s] = color;
	s += NB_THREAD;
	}
    }

__device__
void copyGMtoSM(Sphere* tabSpheresSM, Sphere* ptrDevSpheresGM, int n)
    {
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD_BLOCK = Indice2D::nbThreadBlock();

    int s = TID_LOCAL;

    while (s < n)
	{
	tabSpheresSM[s] = ptrDevSpheresGM[s];
	s += NB_THREAD_BLOCK;
	}
    }

/*----------------------------------------------------------------------*\
|*			End	                    *|
 \*---------------------------------------------------------------------*/
