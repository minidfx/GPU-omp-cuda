#include "hip/hip_runtime.h"
#include "cudaType.h"

#include "Indice2D.h"
#include "IndiceTools.h"
#include "ColorTools.h"
#include "HeatTransfertMathAdvanced.h"

__global__ void diffuseAdvanced(float* ptrDevImageInput, float* ptrDevImageOutput, unsigned int width, unsigned int height, float propagationSpeed);
__global__ void crushAdvanced(float* ptrDevImageHeater, float* ptrDevImage, unsigned int size);
__global__ void displayAdvanced(float* ptrDevImage, uchar4* ptrDevPixels, unsigned int size);

__global__ void diffuseAdvanced(float* ptrDevImageInput, float* ptrDevImageOutput, unsigned int width, unsigned int height, float propagationSpeed)
{
  // Calucul threads available
  const int NB_THREADS = Indice2D::nbThread();
  const int TID = Indice2D::tid();

  // Init service and variable required
  HeatTransfertMathAdvanced math;
  unsigned int totalPixels = width * height;
  unsigned int s = TID;

  while (s < totalPixels)
  {
    int i, j;
    IndiceTools::toIJ(s, width, &i, &j);

    if (i > 0 && i < (height - 1) && j > 0 && j < (width - 1))
    {
      float neighborsHeat[4];
      neighborsHeat[0] = ptrDevImageInput[IndiceTools::toS(width, i - 1, j)];
      neighborsHeat[1] = ptrDevImageInput[IndiceTools::toS(width, i + 1, j)];
      neighborsHeat[2] = ptrDevImageInput[IndiceTools::toS(width, i, j - 1)];
      neighborsHeat[3] = ptrDevImageInput[IndiceTools::toS(width, i, j + 1)];

      ptrDevImageOutput[s] = math.computeHeat(ptrDevImageInput[s], neighborsHeat, 4, propagationSpeed);
    }
    else
    {
      ptrDevImageOutput[s] = ptrDevImageInput[s];
    }

    s += NB_THREADS;
  }
}

__global__ void crushAdvanced(float* ptrDevImageHeater, float* ptrDevImage, unsigned int size)
{
  const int NB_THREADS = Indice2D::nbThread();
  const int TID = Indice2D::tid();

  unsigned int s = TID;
  while (s < size)
  {
    if (ptrDevImageHeater[s] > 0.0)
    {
      ptrDevImage[s] = ptrDevImageHeater[s];
    }

    s += NB_THREADS;
  }
}

__global__ void displayAdvanced(float* ptrDevImage, uchar4* ptrDevPixels, unsigned int size)
    {
    const int NB_THREADS = Indice2D::nbThread();
    const int TID = Indice2D::tid();

    unsigned int s = TID;
    while (s < size)
	{

	float hue = 0.7 - ptrDevImage[s] * 0.7;
	ColorTools::HSB_TO_RVB(hue, 1, 1, &ptrDevPixels[s]);
	ptrDevPixels[s].w = 255;

	s += NB_THREADS;
	}
    }
