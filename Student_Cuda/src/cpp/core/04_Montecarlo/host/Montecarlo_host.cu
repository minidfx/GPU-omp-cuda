#include "hip/hip_runtime.h"
#include <iostream>

#include "cudaTools.h"
#include "Device.h"
#include "curandTools.h"
#include "Montecarlo_host.h"

extern __global__ void computePIWithMonteCarlo(int* ptrDevResult, hiprandState* ptrDevTabGenerators, int nbGen);

bool isMontecarlo()
    {
    const int NB_BLOCS = 256;
    const int NB_THREADS_BY_BLOCK = 1024;
    const int NB_GENERATIONS = 999999;

    dim3 dg(NB_BLOCS, 1, 1);
    dim3 db(NB_THREADS_BY_BLOCK, 1, 1);

    Device::assertDim(dg, db);

    int nbPointsOnIntegralArea;
    int* ptrDevResult;
    hiprandState* ptrDevTabGenerators;

    HANDLE_ERROR(hipMalloc(&ptrDevResult, sizeof(int)));
    HANDLE_ERROR(hipMalloc(&ptrDevTabGenerators, NB_BLOCS * NB_THREADS_BY_BLOCK * sizeof(hiprandState)));
    HANDLE_ERROR(hipMemset(ptrDevResult, 0, sizeof(int)));

    setup_kernel_rand<<<dg,db>>>(ptrDevTabGenerators, 0);
    computePIWithMonteCarlo<<<dg,db>>>(ptrDevResult, ptrDevTabGenerators, NB_GENERATIONS);

    HANDLE_ERROR(hipMemcpy(&nbPointsOnIntegralArea, ptrDevResult, sizeof(int), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(ptrDevResult));
    HANDLE_ERROR(hipFree(ptrDevTabGenerators));

    float piValue = nbPointsOnIntegralArea * 4.0 / NB_GENERATIONS;

    std::cout << "PI = " << piValue << " (Montecarlo)" << std::endl;

    return abs(piValue - 3.141592653589793f) < 0.001;
    }

bool isMontecarloMGPU()
    {
    const int NB_BLOCS = 128;
    const int NB_THREADS_BY_BLOCK = 512;
    const int NB_GENERATIONS_BY_DEVICE = 1000000;

    int nbDevice = Device::getDeviceCount();

    int deviceSums[nbDevice];

#pragma omp parallel for
    for (int deviceID = 0; deviceID < nbDevice; deviceID++)
	{
	HANDLE_ERROR(hipSetDevice(deviceID));

	dim3 dg(NB_BLOCS, 1, 1);
	dim3 db(NB_THREADS_BY_BLOCK, 1, 1);

	Device::assertDim(dg, db);

	int* ptrDevResult;
	hiprandState* ptrDevTabGenerators;

	HANDLE_ERROR(hipMalloc(&ptrDevResult, sizeof(int)));
	HANDLE_ERROR(hipMalloc(&ptrDevTabGenerators, NB_BLOCS * NB_THREADS_BY_BLOCK * sizeof(hiprandState)));
	HANDLE_ERROR(hipMemset(ptrDevResult, 0, sizeof(int)));

	setup_kernel_rand<<<dg,db>>>(ptrDevTabGenerators, deviceID);
	computePIWithMonteCarlo<<<dg,db>>>(ptrDevResult, ptrDevTabGenerators, NB_GENERATIONS_BY_DEVICE);

    	HANDLE_ERROR(hipMemcpy(&deviceSums[deviceID], ptrDevResult, sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(ptrDevResult));
	HANDLE_ERROR(hipFree(ptrDevTabGenerators));
	}

    int finalSum = 0;
    for (int i = 0; i < nbDevice; i++)
	{
	finalSum += deviceSums[0];
	}

    float piValue = finalSum * 4.0 / (nbDevice * NB_GENERATIONS_BY_DEVICE);

    std::cout << "PI = " << piValue << " (Montecarlo Multi GPU)" << std::endl;

    return abs(piValue - 3.141592653589793f) < 0.001;
    }
